#include "hip/hip_runtime.h"
#include "kernel.h"
#include "timer.h"
#include <>

/**
* 自定义核函数，在这个函数里面实现对每个像素的访问
*/
__global__ void kernel(const PtrStepSz<uchar3> src,PtrStep<uchar3> dst)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; //行坐标
    int j = threadIdx.y + blockIdx.y * blockDim.y; //列坐标

//    if(i>50){
//        return;
//    }
    if(i==15 && j==25){
        uchar3 vv = src(i,j);
        printf("(%u,%u,%u)",vv.x,vv.y,vv.z);
    }
    if(i < src.rows && j < src.cols)
    {
        uchar3 v = src(i,j);
        dst(i,j) = make_uchar3(v.y,v.x,v.z);    //紫红色
        //dst(i,j) = make_uchar3(v.x,v.z,v.y);    //浅绿色
        //dst(i,j) = make_uchar3(v.y,v.z,v.x);
    }
}

/**
* 完成核函数的调用
*/
void kernelCaller(const PtrStepSz<uchar3>& src,PtrStep<uchar3> dst)
{
    dim3 block(32,32);
    dim3 grid((src.rows + block.y - 1)/block.y,(src.cols + block.x - 1)/block.x);

    kernel<<<grid,block,0>>>(src,dst);
}

/**
* 外部函数调用这个函数(留给外部的唯一接口)，把GpuMat传进来
*/
void callKernel(const GpuMat& src,GpuMat& dst)
{
    CV_Assert(src.type() == CV_8UC3);
    dst.create(src.size(),src.type());
    kernelCaller(src,dst);
}
