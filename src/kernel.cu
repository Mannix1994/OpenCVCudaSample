#include "hip/hip_runtime.h"
#include "kernel.h"
#include "timer.h"
#include <>

/**
* 自定义核函数，在这个函数里面实现对每个像素的访问
*/
__global__ void kernel(const PtrStepSz<uchar3> src,PtrStep<uchar3> dst)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; //列坐标
    int j = threadIdx.y + blockIdx.y * blockDim.y; //行坐标

//    if(i>50){
//        return;
//    }
    if(i==25 && j==15){
        uchar3 vv = src(j,i);
        printf("(%u,%u,%u)",vv.x,vv.y,vv.z);
    }
    if(j < src.rows && i < src.cols)
    {
        uchar3 v = src(j,i);
        dst(j,i) = make_uchar3(v.y,v.x,v.z);    //紫红色
    }
}

/**
* 完成核函数的调用
*/
void kernelCaller(const PtrStepSz<uchar3>& src,PtrStep<uchar3> dst)
{
    dim3 block(32,//一个block有多少列
            32);//一个block多少行
    dim3 grid((src.cols + block.x - 1)/block.x,// 列的方向的block数目
            (src.rows + block.y - 1)/block.y);// 行的方向的block数目

    kernel<<<grid,block,0>>>(src,dst);
}

/**
* 外部函数调用这个函数(留给外部的唯一接口)，把GpuMat传进来
*/
void callKernel(const GpuMat& src,GpuMat& dst)
{
    CV_Assert(src.type() == CV_8UC3);
    dst.create(src.size(),src.type());
    kernelCaller(src,dst);
}
