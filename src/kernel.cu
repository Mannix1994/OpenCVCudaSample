#include "hip/hip_runtime.h"
#include "kernel.h"
#include "timer.h"
#include <>

/**
* 自定义核函数，在这个函数里面实现对每个像素的访问
*/
__global__ void kernel(const PtrStepSz<uchar3> src,PtrStep<uchar3> dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    uchar3 v = src(0,0);
    if(x==0 && y==0){
        printf("(%u,%u,%u)",v.x,v.y,v.z);
    }
    if(x < src.cols && y < src.rows)
    {
        uchar3 v = src(y,x);
        dst(y,x) = make_uchar3(v.y,v.x,v.z);    //紫红色
        //dst(y,x) = make_uchar3(v.x,v.z,v.y);    //浅绿色
        //dst(y,x) = make_uchar3(v.y,v.z,v.x);
    }
}

/**
* 完成核函数的调用
*/
void kernelCaller(const PtrStepSz<uchar3>& src,PtrStep<uchar3> dst)
{
    dim3 block(32,32);
    dim3 grid((src.cols + block.x - 1)/block.x,(src.rows + block.y - 1)/block.y);

    kernel<<<grid,block,0>>>(src,dst);
//    kernel<<<1024,1>>>(src,dst);
}

/**
* 外部函数调用这个函数(留给外部的唯一接口)，把GpuMat传进来
*/
void callKernel(const GpuMat& src,GpuMat& dst,Stream& stream)
{
    CV_Assert(src.type() == CV_8UC3);
    dst.create(src.size(),src.type());
    kernelCaller(src,dst);
}
