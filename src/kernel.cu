#include "hip/hip_runtime.h"
#include "kernel.h"
#include "timer.h"
#include <>

/**
* 自定义核函数，在这个函数里面实现对每个像素的访问
*/
__global__ void kernel(const PtrStepSz<uchar3> src,PtrStep<uchar3> dst)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; //行坐标
    int j = threadIdx.y + blockIdx.y * blockDim.y; //列坐标

//    if(i>50){
//        return;
//    }
    uchar3 vv = src(0,0);
    if(i==0 && j==0){
        printf("(%u,%u,%u)",vv.x,vv.y,vv.z);
    }
    if(i < src.cols && j < src.rows)
    {
        uchar3 v = src(i,j);
        dst(i,j) = make_uchar3(v.y,v.x,v.z);    //紫红色
        //dst(i,j) = make_uchar3(v.x,v.z,v.y);    //浅绿色
        //dst(i,j) = make_uchar3(v.y,v.z,v.x);
    }
}

/**
* 完成核函数的调用
*/
void kernelCaller(const PtrStepSz<uchar3>& src,PtrStep<uchar3> dst)
{
    dim3 block(32,32);
    dim3 grid((src.cols + block.x - 1)/block.x,(src.rows + block.y - 1)/block.y);

    kernel<<<grid,block,0>>>(src,dst);
//    kernel<<<1024,1>>>(src,dst);
}

/**
* 外部函数调用这个函数(留给外部的唯一接口)，把GpuMat传进来
*/
void callKernel(const GpuMat& src,GpuMat& dst,Stream& stream)
{
    CV_Assert(src.type() == CV_8UC3);
    dst.create(src.size(),src.type());
    kernelCaller(src,dst);
}
