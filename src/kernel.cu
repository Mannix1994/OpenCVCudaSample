#include "hip/hip_runtime.h"
#include "kernel.h"
#include "timer.h"

/**
* 自定义核函数，在这个函数里面实现对每个像素的访问
*/
__global__ void kernel(const PtrStepSz<uchar3> src,PtrStep<uchar3> dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if(x < src.cols && y < src.rows)
    {
        uchar3 v = src(y,x);
        dst(y,x) = make_uchar3(v.y,v.x,v.z);    //紫红色
        //dst(y,x) = make_uchar3(v.x,v.z,v.y);    //浅绿色
        //dst(y,x) = make_uchar3(v.y,v.z,v.x);
    }
}

/**
* 完成核函数的调用
*/
void kernelCaller(const PtrStepSz<uchar3>& src,PtrStep<uchar3> dst,hipStream_t stream)
{
    dim3 block(32,32);
    dim3 grid((src.cols + block.x - 1)/block.x,(src.rows + block.y - 1)/block.y);

    kernel<<<grid,block,0,stream>>>(src,dst);
//    kernel<<<1024,1>>>(src,dst);
    if(stream == 0)
        hipDeviceSynchronize();
}

/**
* 外部函数调用这个函数(留给外部的唯一接口)，把GpuMat传进来
*/
void callKernel(const GpuMat& src,GpuMat& dst,Stream& stream)
{
    CV_Assert(src.type() == CV_8UC3);
    dst.create(src.size(),src.type());
    hipStream_t s = StreamAccessor::getStream(stream);
    kernelCaller(src,dst,s);
}
